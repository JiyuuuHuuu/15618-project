#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include "kernel.h"
#include "hip/hip_vector_types.h"
#include "helper.cu_inl"
#include "pattern.cu_inl"
#include "color.cu_inl"
#define TX 32
#define TY 32

/*
  particle array layout
  each cell is a struct firework;
  NOTE: each firework consists of multiple particle
  | |: empty, |*|: occupied
  | | |buffer_tail *|*|*|*|*|buffer_head| | |
*/

__constant__ uchar4 cuPalette[256];
__constant__ particle cuSchedule[MAX_SCHEDULE_NUM];

__device__
void launchSchedule(particle *particles, int *schedule_idx, int *buffer_head, float t) {
  // check and copy firework from schedule to work buffer for display
  firework *buffer = reinterpret_cast<firework *>(particles);
  const int blk_idx = blockIdx.y*gridDim.x + blockIdx.x;
  const int thd_idx = threadIdx.y*blockDim.x + threadIdx.x;
  const int idx = blk_idx*blockDim.x*blockDim.y + thd_idx; // get 1D index
  const int schedule_idx_local = *schedule_idx;
  const int buffer_head_local = *buffer_head;
  __syncthreads();
  if (idx + schedule_idx_local >= MAX_SCHEDULE_NUM ||
      idx + buffer_head_local >= MAX_PARTICLE_NUM) return;

  particle schedule_particle = cuSchedule[idx + schedule_idx_local];
  if (schedule_particle.t_0 >= 0.0f && schedule_particle.t_0 <= t) {
    printf("move to display\n");
    buffer[idx + buffer_head_local].pack[0] = schedule_particle;

    // update indices
    if (idx + schedule_idx_local + 1 == MAX_SCHEDULE_NUM ||
        cuSchedule[idx + schedule_idx_local + 1].t_0 > t) {
      *schedule_idx += idx + 1;
      *buffer_head += idx + 1;
    } else if (cuSchedule[idx + schedule_idx_local + 1].t_0 < 0) {
      // no schedule left to display
      *schedule_idx = MAX_SCHEDULE_NUM;
      *buffer_head += idx + 1;
    }
  }
}

__device__
void updateParticle(particle *particles, int *schedule_idx, int *buffer_head, int *buffer_tail, float t) {
  firework *buffer = reinterpret_cast<firework *>(particles);
  const int blk_idx = blockIdx.y*gridDim.x + blockIdx.x;
  const int thd_idx = threadIdx.y*blockDim.x + threadIdx.x;
  const int idx = blk_idx*blockDim.x*blockDim.y + thd_idx; // get 1D index
  const int firework_per_it = (gridDim.x*gridDim.y*blockDim.x*blockDim.y)/PARTICLE_NUM_PER_FIREWORK;
  const int particle_idx = idx % PARTICLE_NUM_PER_FIREWORK;

  unsigned int seed = idx*7 + (unsigned int)(t*100);
  int buffer_idx = idx/PARTICLE_NUM_PER_FIREWORK + *buffer_tail;
  for (int i = buffer_idx; i < *buffer_head; i += firework_per_it) {
    firework curr_firework = buffer[i];
    particle upshoot = curr_firework.pack[0];
    particle curr = curr_firework.pack[particle_idx];
    if (upshoot.t_0 < 0) continue;
    if (particle_idx == 0) {
      // upshooting particle
      if (curr.explosion_height > 0) {
        float2 p = currP(curr.p_0, curr.v_0, curr.a, t - curr.t_0);
        if (p.y <= curr.explosion_height) curr.explosion_height = -1.0f; // mark explosion phase
      } else {
        int isEnd = 1;
        for (int j = 1; j < PARTICLE_NUM_PER_FIREWORK; j++) {
          if (curr_firework.pack[j].t_0 >= 0) {
            isEnd = 0;
            break;
          }
        }
        if (isEnd) curr.t_0 = -1.0f; // mark this firework as evicted
      }
    } else {
      // child particle
      if (upshoot.explosion_height > 0) {
        float2 p_up = currP(upshoot.p_0, upshoot.v_0, upshoot.a, t - upshoot.t_0);
        if (p_up.y <= upshoot.explosion_height) {
          // patternArray[upshoot.color](curr, p_up, t, particle_idx, seed);
          patterns(curr, p_up, t, particle_idx, seed, upshoot.color);
        }
      } else {
        // check particle end
        if (t - curr.t_0 >= curr.explosion_height)
          curr.t_0 = -1.0f;
      }
    }
    __syncthreads();
    buffer[i].pack[particle_idx] = curr;
  }

  // TODO: implement circular allocation
}

__global__
void fireworkKernel(uchar4 *d_out, int w, int h, particle *particles, tail *tails, float t, int *buffer_head, int *buffer_tail, int *schedule_idx) {
  const int c = blockIdx.x*blockDim.x + threadIdx.x;
  const int r = blockIdx.y*blockDim.y + threadIdx.y;
  const int idx = c + r*w; // 1D indexing
  launchSchedule(particles, schedule_idx, buffer_head, t);
  __syncthreads();

  // display
  int tail_increment = 0;
  int freeup = 1;
  firework *buffer = reinterpret_cast<firework *>(particles);
  
  float2 pixel_pos = make_float2((float)c, (float)r);
  if (!((c >= w) || (r >= h))) {
    tail curr_tail = tails[idx];
    uchar4 pixel_color = make_uchar4(0, 0, 0, 255);
    for (int i = *buffer_tail; i < *buffer_head; i++) {
      firework *curr_firework = buffer + i;
      particle upshoot = curr_firework->pack[0];
      if (upshoot.t_0 < 0) {
        if (freeup) tail_increment++;
        continue;
      }
      freeup = 0;
      if (upshoot.explosion_height > 0) {
        // only upshooting particle need display

        // float2 p = currP(upshoot.p_0, upshoot.v_0, upshoot.a, t - upshoot.t_0);
        // if (isWithinDistance(p, pixel_pos, upshoot.r)) {
        //   pixel_color = cuPalette[upshoot.color]; // TODO: support particle overlap
        // }
        upshoots(pixel_color, t, 0, upshoot, pixel_pos, curr_tail);
      } else {
        // firework after explosion
        for (int j = 1; j < PARTICLE_NUM_PER_FIREWORK; j++) {
          particle curr = curr_firework->pack[j];
          if (curr.t_0 < 0 || curr.t_0 > t) continue;
            // pixel_color = cuPalette[curr.color]; // TODO: support particle overlap
          colors(pixel_color, t, j, curr, pixel_pos, curr_tail);
        }
      }
    }
    tail_colors(pixel_color, t, 0, curr_tail);

    tails[idx] = curr_tail;
    d_out[idx] = pixel_color;
  }
  if (idx == 0) *buffer_tail += tail_increment;
  __syncthreads();

  updateParticle(particles, schedule_idx, buffer_head, buffer_tail, t);
}

void kernelLauncher(uchar4 *d_out, int w, int h, particle *particles, tail *tails, int *idx_holder, float t) {
  const dim3 blockSize(TX, TY);
  const dim3 gridSize = dim3((w + TX - 1)/TX, (h + TY - 1)/TY);
  fireworkKernel<<<gridSize, blockSize>>>(d_out, w, h, particles, tails, t, idx_holder, idx_holder+1, idx_holder+2);
  hipError_t cudaStatus = hipDeviceSynchronize();
  if (cudaStatus != hipSuccess) {
    printf("CUDA error after hipDeviceSynchronize: %s\n", hipGetErrorString(cudaStatus));
    exit(1);
  }
}

void makePalette(void) {
  uchar4 palette[256];

  palette[0] = make_uchar4(255, 0, 0, 255);
  palette[1] = make_uchar4(0, 255, 0, 255);
  palette[2] = make_uchar4(0, 0, 255, 255);
  palette[3] = make_uchar4(255, 255, 0, 255);
  palette[4] = make_uchar4(255, 0, 255, 255);
  palette[5] = make_uchar4(0, 255, 255, 255);

  hipMemcpyToSymbol(HIP_SYMBOL(cuPalette), palette, sizeof(uchar4) * 256);
}

void setUpSchedule(particle *particles_host) {
  hipMemcpyToSymbol(HIP_SYMBOL(cuSchedule), particles_host, MAX_SCHEDULE_NUM*sizeof(particle));
}