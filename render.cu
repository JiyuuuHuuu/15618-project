#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#include "render.cuh"

__global__ void test_kernel(void) {
}

namespace firework {
	void parallel_render(void) {
		test_kernel <<<1, 1>>> ();
		printf("Hello, world!\n");
	}
}